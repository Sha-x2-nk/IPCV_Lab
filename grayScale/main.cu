#include "hip/hip_runtime.h"
#include<iostream>
#include<time.h>
#include<opencv2/opencv.hpp>
#define BLOCK_SIZE 32 // 32 * 32= 1024. 1024 max threads per block
__global__ void convert2Gray(unsigned char* img_in, unsigned char *img_out, int height, int width){
    int col= threadIdx.x + blockIdx.x*blockDim.x;
    int row= threadIdx.y + blockIdx.y*blockDim.y;

    if(col< width && row< height){
        int grayIdx= row*width + col;
        int rgbIdx= grayIdx*3;
        unsigned char r= img_in[rgbIdx    ];
        unsigned char g= img_in[rgbIdx + 1];
        unsigned char b= img_in[rgbIdx + 2];
        
        img_out[grayIdx]= 0.21f*r + 0.71f*g + 0.07f*b;
    }
    
}

int main(){
    cv::Mat img= cv::imread("./assets/flower.jpeg", cv::IMREAD_COLOR);
    cv::Size s= img.size();
    int height= s.height;
    int width= s.width;
    if(img.empty()){
        std::cout<<"\nERROR LOADING IMAGE.";
        return -1;
    }
    cv::cvtColor(img, img, cv::COLOR_BGR2RGB);
    unsigned char *img_h= img.data;

    unsigned char *img_in_d, *img_out_d;
    hipMalloc((void **)&img_in_d, sizeof(unsigned char)*height*width*3);
    hipMalloc((void **)&img_out_d, sizeof(unsigned char)*height*width);
    hipMemcpy(img_in_d, img_h, sizeof(unsigned char)*height*width*3, hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + BLOCK_SIZE - 1)/BLOCK_SIZE, (height + BLOCK_SIZE - 1)/ BLOCK_SIZE);

    convert2Gray<<<grid, block>>>(img_in_d, img_out_d, height, width);
    hipDeviceSynchronize();

    unsigned char *img_out_h= (unsigned char*)malloc(sizeof(char)*height*width);
    hipMemcpy(img_out_h, img_out_d, sizeof(unsigned char)*height*width, hipMemcpyDeviceToHost);

    cv::Mat grayImg= cv::Mat(height, width, CV_8UC1, (void *)img_out_h);
    cv::imwrite("flower_gray.jpg", grayImg);
    cv::imshow("IMG", grayImg);
    cv::waitKey(0);
    
    return 0;
}
