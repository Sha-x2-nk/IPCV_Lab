#include "hip/hip_runtime.h"
#include<iostream>
#include<opencv2/opencv.hpp>
#define BLOCK_SIZE 32 // 32x32= 1024 - max per block
#define BLUR_SIZE 16 // 16*2 + 1 size k square ka filter
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void blurKernel(unsigned char* img_in, unsigned char* img_out, int img_height, int img_width){
    int row= threadIdx.y + blockDim.y * blockIdx.y;
    int col= threadIdx.x + blockDim.x * blockIdx.x;

    if(row< img_height and col< img_width){
        int pix_val[3]={0};
        int pix_num= 0;
        for(int rowShift= -BLUR_SIZE; rowShift<= BLUR_SIZE; ++rowShift){
            for(int colShift= -BLUR_SIZE; colShift<= BLUR_SIZE; ++colShift){
                int curRow= row + rowShift; 
                int curCol= col + colShift;
                if(curRow< img_height and curCol< img_width){
                    int curIdx= (img_width*curRow + curCol)*3;
                    for(int ch= 0; ch< 3; ++ch){
                        pix_val[ch]+= img_in[curIdx + ch];
                    }
                    ++pix_num;
                }
            }
        }
        int outIdx= (row*img_width + col)*3;
        for(int ch= 0; ch< 3; ++ch){
            pix_val[ch]/= pix_num;
            img_out[outIdx + ch]= pix_val[ch];
        }
    }
}

int main(){
    cv::Mat img= cv::imread("assets/flower.jpeg", cv::IMREAD_COLOR); // opencv BGR
    cv::cvtColor(img, img, cv::COLOR_BGR2RGB);
    cv::Size img_sz= img.size();
    int img_height= img_sz.height, img_width= img_sz.width;
    int arr_size= sizeof(unsigned char)*img_height*img_width*3;

    unsigned char *img_in_h, *img_out_h, *img_in_d, *img_out_d;
    img_in_h= img.data;
    img_out_h= (unsigned char*)malloc(arr_size);
    

    // allocate cuda memory
    hipMalloc((void **)&img_in_d, arr_size);
    hipMalloc((void **)&img_out_d, arr_size);
    // cudaCheckErrors("CUDA ERROR WHILE ALLOCATION MEM.");

    hipMemcpy(img_in_d, img_in_h, arr_size, hipMemcpyHostToDevice);
    // cudaCheckErrors("CUDA ERROR COPYING FROM HOST TO DEVICE.");

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((img_width + BLOCK_SIZE - 1)/BLOCK_SIZE, (img_height + BLOCK_SIZE - 1)/BLOCK_SIZE);

    blurKernel<<<grid, block>>>(img_in_d, img_out_d, img_height, img_width);
    hipDeviceSynchronize();
    
    hipMemcpy(img_out_h, img_out_d, arr_size, hipMemcpyDeviceToHost);
    // cudaCheckErrors("CUDA ERROR COPYING FROM DEVICE TO HOST.");

    cv::Mat img_res= cv::Mat(img_height, img_width, CV_8UC3, (void *)img_out_h);
    cv::imshow("IMG", img_res);
    cv::waitKey(0);
    
    
    return 0;
}